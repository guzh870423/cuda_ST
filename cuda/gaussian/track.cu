#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <gsl/gsl_rng.h>
#include <gsl/gsl_randist.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "Pass.h"
#include "constants.h"
#include "lattice_PAR.h"



using namespace std;



void emittance(COORD *y, REAL *PEx, REAL *PEy, REAL *PEdelta)
{

	REAL avg_x=0,avg_xp=0,avg_y=0,avg_yp=0,avg_delta=0,sig_xx=0,sig_xpxp=0,sig_xxp=0,sig_yy=0,sig_ypyp=0,sig_yyp=0,sig_delta=0;
	for(int i=0;i<_Npart;i++)
	{
		avg_x+=y[i].x[x_]/_Npart;
		avg_xp+=y[i].x[px_]/(1+y[i].x[delta_])/_Npart;
		avg_y+=y[i].x[y_]/_Npart;
		avg_yp+=y[i].x[py_]/(1+y[i].x[delta_])/_Npart;
		avg_delta+=y[i].x[delta_]/_Npart;
	}

	for(int i=0;i<_Npart;i++)
	{
		sig_xx+=(y[i].x[x_]-avg_x)*(y[i].x[x_]-avg_x)/_Npart;
		sig_xpxp+=(y[i].x[px_]/(1+y[i].x[delta_])-avg_xp)*(y[i].x[px_]/(1+y[i].x[delta_])-avg_xp)/_Npart;
		sig_xxp+=(y[i].x[x_]-avg_x)*(y[i].x[px_]/(1+y[i].x[delta_])-avg_xp)/_Npart;
		sig_yy+=(y[i].x[y_]-avg_y)*(y[i].x[y_]-avg_y)/_Npart;
		sig_ypyp+=(y[i].x[py_]/(1+y[i].x[delta_])-avg_yp)*(y[i].x[py_]/(1+y[i].x[delta_])-avg_yp)/_Npart;
		sig_yyp+=(y[i].x[y_]-avg_y)*(y[i].x[py_]/(1+y[i].x[delta_])-avg_yp)/_Npart;
		sig_delta+=(y[i].x[delta_]-avg_delta)*(y[i].x[delta_]-avg_delta)/_Npart;
	}
	
	*PEx=sqrt(sig_xx*sig_xpxp-sig_xxp*sig_xxp);
	*PEy=sqrt(sig_yy*sig_ypyp-sig_yyp*sig_yyp);
	*PEdelta=sqrt(sig_delta);
}
int main(int argc, char** argv)
{




//initialization
  	const gsl_rng_type * T;
  	gsl_rng * r;

  	gsl_rng_env_setup();

  	T = gsl_rng_default;
 	r = gsl_rng_alloc (T);

  	COORD *part, *dpart;
	int size = _Npart * sizeof(COORD);
	part=(COORD*)malloc(size);
	hipMalloc(&dpart,size);
	
	
	REAL phi_x,phi_y,Jx,Jy,Ex,Ey,Sdelta;
	int i,n;

	for(i=0;i<_Npart;i++)
	{
	     
	     do {Jx=gsl_ran_exponential(r, 2*E_x);}
		while(Jx>E_x*6);
	     do {Jy=gsl_ran_exponential(r, 2*E_y);}
		while(Jy>E_y*6);
		phi_x=gsl_ran_flat(r,0,2*M_PI);
		phi_y=gsl_ran_flat(r,0,2*M_PI);

		part[i].x[x_]=sqrt(Jx*Beta_x)*cos(phi_x);
		part[i].x[px_]=sqrt(Jx/Beta_x)*sin(phi_x);
		part[i].x[y_]=sqrt(Jy*Beta_y)*cos(phi_y);
		part[i].x[py_]=sqrt(Jy/Beta_y)*sin(phi_y);
		part[i].x[z_]=0;
		part[i].x[delta_]=0.00;			
	
	}
//	part[0].x[0]=0.000;part[0].x[1]=0.000;part[0].x[2]=0.000;part[0].x[3]=0.000;part[0].x[5]=0.00;


	hipMemcpy(dpart,part,size,hipMemcpyHostToDevice);

       ofstream outfile("abc.txt");
        outfile.close();
	for(n=0;n<_Nturn1;n++)
	{
		Track<<<_BlockNum,_ThreadNum>>>(dpart,n);

		hipMemcpy(part,dpart,size,hipMemcpyDeviceToHost);
	emittance(part,&Ex,&Ey,&Sdelta);

       ofstream outfile("abc.txt",ios::app);
     outfile<<n<<"    "<<Ex<<"    "<<Ey<<"    "<<Sdelta<<endl;
 //     outfile<<part[0].x[0]<<"  "<<part[0].x[1]<<"  "<<part[0].x[2]<<"  "<<part[0].x[3]<<"  "<<part[0].x[4]<<"  "<<part[0].x[5]<<endl;
//	for(int k=0;k<_Npart;k++) {  if(abs(part[k].x[0])>10||abs(part[k].x[1])>10||part[k].x[2]>10||part[k].x[3]>10||part[k].x[5]>2) {cout<<n<<"  "<<part[k].x[0]<<"  "<<part[k].x[1]<<"  "<<part[k].x[2]<<"  "<<part[k].x[3]<<"  "<<part[k].x[4]<<"  "<<part[k].x[5]<<endl;}  }
        outfile.close();

	}


	cout<<part[0].x[0]<<"  "<<part[0].x[1]<<"  "<<part[0].x[2]<<"  "<<part[0].x[3]<<"  "<<part[0].x[4]<<"  "<<part[0].x[5]<<endl;
 

	free(part);
	hipFree(dpart);

	gsl_rng_free (r);

}

