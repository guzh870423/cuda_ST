#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <gsl/gsl_rng.h>
#include <gsl/gsl_randist.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "time.h"
#include "Pass.h"
#include "constants.h"
#include "lattice_PAR.h"



using namespace std;



void emittance(COORD *y, REAL *PEx, REAL *PEy, REAL *PEdelta)
{

	REAL avg_x=0,avg_xp=0,avg_y=0,avg_yp=0,avg_delta=0,sig_xx=0,sig_xpxp=0,sig_xxp=0,sig_yy=0,sig_ypyp=0,sig_yyp=0,sig_delta=0;
	for(int i=0;i<_Npart;i++)
	{
		avg_x+=y[i].x[x_]/_Npart;
		avg_xp+=y[i].x[px_]/(1+y[i].x[delta_])/_Npart;
		avg_y+=y[i].x[y_]/_Npart;
		avg_yp+=y[i].x[py_]/(1+y[i].x[delta_])/_Npart;
		avg_delta+=y[i].x[delta_]/_Npart;
	}

	for(int i=0;i<_Npart;i++)
	{
		sig_xx+=(y[i].x[x_]-avg_x)*(y[i].x[x_]-avg_x)/_Npart;
		sig_xpxp+=(y[i].x[px_]/(1+y[i].x[delta_])-avg_xp)*(y[i].x[px_]/(1+y[i].x[delta_])-avg_xp)/_Npart;
		sig_xxp+=(y[i].x[x_]-avg_x)*(y[i].x[px_]/(1+y[i].x[delta_])-avg_xp)/_Npart;
		sig_yy+=(y[i].x[y_]-avg_y)*(y[i].x[y_]-avg_y)/_Npart;
		sig_ypyp+=(y[i].x[py_]/(1+y[i].x[delta_])-avg_yp)*(y[i].x[py_]/(1+y[i].x[delta_])-avg_yp)/_Npart;
		sig_yyp+=(y[i].x[y_]-avg_y)*(y[i].x[py_]/(1+y[i].x[delta_])-avg_yp)/_Npart;
		sig_delta+=(y[i].x[delta_]-avg_delta)*(y[i].x[delta_]-avg_delta)/_Npart;
	}
	
	*PEx=sqrt(sig_xx*sig_xpxp-sig_xxp*sig_xxp);
	*PEy=sqrt(sig_yy*sig_ypyp-sig_yyp*sig_yyp);
	*PEdelta=sqrt(sig_delta);
}
int main(int argc, char** argv)
{

  clock_t start, finish;
	start = clock();
//get a bunch of random numbers
	REAL *queue, *dqueue;
	queue=(REAL*)malloc(_pool*sizeof(REAL));
	hipMalloc(&dqueue,_pool*sizeof(REAL));

	ifstream infile1("queue");
	for(int i1=0;i1<_pool;i1++) 	infile1>>queue[i1];
 	infile1.close();
	hipMemcpy(dqueue,queue,_pool*sizeof(REAL),hipMemcpyHostToDevice);
	free(queue);
//initialization
  	const gsl_rng_type * T;
  	gsl_rng * r;

  	gsl_rng_env_setup();

  	T = gsl_rng_default;
 	r = gsl_rng_alloc (T);

  	COORD *part, *dpart;
	int size = _Npart * sizeof(COORD);
hipHostAlloc( (void**)&part,size,hipHostMallocDefault );
//	part=(COORD*)malloc(size);
	hipMalloc(&dpart,size);
	
	
	REAL phi_x,phi_y,Jx,Jy,Ex,Ey,Sdelta;
	int i,n;

	for(i=0;i<_Npart;i++)
	{
	     
	     do {Jx=gsl_ran_exponential(r, 2*E_x);}
		while(Jx>E_x*180);
	     do {Jy=gsl_ran_exponential(r, 2*E_y);}
		while(Jy>E_y*180);
		phi_x=gsl_ran_flat(r,0,2*M_PI);
		phi_y=gsl_ran_flat(r,0,2*M_PI);

		part[i].x[x_]=sqrt(Jx*Beta_x)*cos(phi_x);
		part[i].x[px_]=sqrt(Jx/Beta_x)*sin(phi_x);
		part[i].x[y_]=sqrt(Jy*Beta_y)*cos(phi_y);
		part[i].x[py_]=sqrt(Jy/Beta_y)*sin(phi_y);
		part[i].x[z_]=0;
		part[i].x[delta_]=0.00;			
	
	}
//	part[0].x[0]=0.000;part[0].x[1]=0.000;part[0].x[2]=0.000;part[0].x[3]=0.000;part[0].x[5]=0.01;


	hipMemcpy(dpart,part,size,hipMemcpyHostToDevice);
//Initiate lattice
	Initiate_lattice();

       ofstream outfile("abc.txt");
        outfile.close();
	for(n=0;n<_Nturn1;n++)
	{
		Track<<<_BlockNum,_ThreadNum>>>(dpart,dqueue,n);

		hipMemcpy(part,dpart,size,hipMemcpyDeviceToHost);
	emittance(part,&Ex,&Ey,&Sdelta);

       ofstream outfile("abc.txt",ios::app);
     outfile<<setw(4)<<n*_Nturn2<<scientific<<setw(15)<<Ex<<scientific<<setw(15)<<Ey<<setw(15)<<scientific<<Sdelta<<endl;
 //     outfile<<part[0].x[0]<<"  "<<part[0].x[1]<<"  "<<part[0].x[2]<<"  "<<part[0].x[3]<<"  "<<part[0].x[4]<<"  "<<part[0].x[5]<<endl;
//	for(int k=0;k<_Npart;k++) {  if(abs(part[k].x[0])>10||abs(part[k].x[1])>10||part[k].x[2]>10||part[k].x[3]>10||part[k].x[5]>2) {cout<<n<<"  "<<part[k].x[0]<<"  "<<part[k].x[1]<<"  "<<part[k].x[2]<<"  "<<part[k].x[3]<<"  "<<part[k].x[4]<<"  "<<part[k].x[5]<<endl;}  }
        outfile.close();
/*		Jx=part[0].x[0]*part[0].x[0]/Beta_x + part[0].x[1]*part[0].x[1]*Beta_x;
		Jy=part[0].x[2]*part[0].x[2]/Beta_y + part[0].x[3]*part[0].x[3]*Beta_y; REAL J2=Jx+2*Jy;
		phi_x=acos( part[i].x[0]/sqrt(Jx*Beta_x));
unsigned num=(int) phi_x;if(num>10) phi_x=0;
		phi_y=acos( part[i].x[2]/sqrt(Jy*Beta_y));
num=(int) phi_y;if(num>10) phi_y=0;
		REAL Hamilt=sqrt(Jy)*Jx * abs( cos(2*phi_x-phi_y) ) / (sqrt(J2)*J2); //scaled Hamiltonian
cout<<J2<<endl; */
	}

//particles' distribution
	const int Nbin = 100;
	unsigned int binx[Nbin+1]={0},biny[Nbin+1]={0},num;
	REAL phix,phiy,Hamilt,J2;
 //     ofstream outfile2("particles_action");

	for(i=0;i<_Npart;++i)
	{
		Jx=part[i].x[0]*part[i].x[0]/Beta_x + part[i].x[1]*part[i].x[1]*Beta_x;
		Jy=part[i].x[2]*part[i].x[2]/Beta_y + part[i].x[3]*part[i].x[3]*Beta_y;
		J2=Jx+2*Jy;
		phix=atan2( part[i].x[1]*sqrt(Beta_x),part[i].x[0]/sqrt(Beta_x));
//num=(int) phix;if(num>10) phix=0;
		phiy=atan2( part[i].x[3]*sqrt(Beta_y),part[i].x[2]/sqrt(Beta_y));
//num=(int) phiy;if(num>10) phiy=0;
		Hamilt=sqrt(Jy)*Jx * abs( cos(2*phix-phiy) ) / (sqrt(J2)*J2); //scaled Hamiltonian
//		outfile2<<setw(15)<<scientific<<Jx<<setw(15)<<scientific<<Jy<<endl;
		if(Jx>=10*Ex)
		{
			++binx[Nbin];
		}
		else
		{
			
			num=(int) (Jx/ (10*Ex/Nbin));

			++binx[num];
		}
		if(Jy>=10*Ey)
		{
			++biny[Nbin];
		}
		else
		{
			num=(int) (Jy/ (10*Ey/Nbin));
			++biny[num];
		}

/*
		if(Hamilt>=0.3)
		{
			++binx[Nbin];
		}
		else
		{
			
			num=(int) (Hamilt/ (0.3/Nbin));
			++binx[num];
		}

		if( (Jx+2*Jy)/(Ex+2*Ey)>=10.0)
		{
			++biny[Nbin];
		}
		else
		{
			
			num=(int) ((Jx+2*Jy)/(Ex+2*Ey)/ (10.0/Nbin));
			++biny[num];
		}
*/
	}
//	outfile2.close();
//histogram
	ofstream outfile3("histo");
	for(i=0;i<Nbin;++i)
	{
		outfile3<<setw(15)<<10.0/Nbin*(i+0.5)<<setw(6)<<binx[i]<<setw(15)<<10.0/Nbin*(i+0.5)<<setw(6)<<biny[i]<<endl;

//		outfile3<<setw(15)<<0.3/Nbin*(i+0.5)<<setw(6)<<binx[i]<<setw(15)<<10.0/Nbin*(i+0.5)<<setw(6)<<biny[i]<<endl;
	}
	outfile3.close();

	cout<<part[0].x[0]<<"  "<<part[0].x[1]<<"  "<<part[0].x[2]<<"  "<<part[0].x[3]<<"  "<<part[0].x[4]<<"  "<<part[0].x[5]<<endl;
 
hipHostFree( part );
//	free(part);
	hipFree(dpart);

	hipFree(dqueue);
	gsl_rng_free (r);

	finish = clock();
	cout<<(finish-start)/CLOCKS_PER_SEC<<" sec"<<endl;
}

