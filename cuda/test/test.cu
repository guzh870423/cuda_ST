#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"
#define BlockNum 10
#define ThreadNum 10

using namespace std;
__global__ void count(float *dnumbers)
{
dnumbers[blockIdx.x*blockDim.x+threadIdx.x]=blockIdx.x*blockDim.x+threadIdx.x;

}
__global__ void add(int a, int b, int *c)
{
  *c = a + b;
}
int main()
{
/*
int size = BlockNum * ThreadNum * sizeof(float);
float *numbers, * dnumbers;

numbers = (float *)malloc(size);
hipMalloc(&dnumbers,size);

 count<<<BlockNum,ThreadNum>>>(dnumbers);
hipMemcpy(numbers,dnumbers,size,hipMemcpyDeviceToHost);
	for(int i=0;i<BlockNum * ThreadNum;++i)
	{
		cout<<numbers[i]<<endl;
	
	}
*/

int c;
int *dev_c;
HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int) ) );
add<<<1,1>>>(2,7,dev_c);
HANDLE_ERROR( hipMemcpy( &c,
dev_c,
sizeof(int),
hipMemcpyDeviceToHost ) );
printf( "2 + 7 = %d\n", c );
hipFree(dev_c);
return 0;
}